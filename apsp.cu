#include <cstdlib>
#pragma once
#include <hip/hip_runtime.h>


namespace apsp {
    typedef unsigned long size_t;
    typedef int T;
#define BS (12)
#define INF (INT_MAX / 2)

    __global__ void Calc_(int n, T *deviceInput) {
        size_t i = blockIdx.x * blockDim.x + threadIdx.x;
        size_t j = blockIdx.y * blockDim.y + threadIdx.y;

        size_t bi = blockIdx.x;
        size_t bj = blockIdx.y;

        size_t ti = threadIdx.x;
        size_t tj = threadIdx.y;

        T res = deviceInput[i * n + j];

        for (size_t b1 = bi * BS * n, b2 = bj * BS; b2 < n * n; b1 += BS, b2 += BS * n) {

            __shared__ T sm1[BS][BS], sm2[BS][BS];

            sm1[ti][tj] = deviceInput[b1 + ti * n + tj];
            sm2[ti][tj] = deviceInput[b2 + ti * n + tj];

            __syncthreads();

            for (size_t k = 0; k < BS; k++) {
                if (sm1[ti][k] + sm2[k][tj] < res) { res = sm1[ti][k] + sm2[k][tj]; }
            }

            __syncthreads();
        }

        deviceInput[i * n + j] = res;
    }

    class FloydWarshall {
        private:
        size_t n;
        T **distance, *deviceInput;

        public:
        FloydWarshall(size_t n_) {
            n = (n_ + BS - 1) / BS * BS;
            distance = (T **) malloc(n * sizeof(T *));
            distance[0] = (T *) malloc(n * n * sizeof(T));

            hipMalloc((void **) &deviceInput, n * n * sizeof(T));

            for (size_t i = 0; i < n; i++) {
                distance[i] = distance[0] + i * n;
                for (size_t j = 0; j < n; j++) { distance[i][j] = INF; }
            }
        }
        ~FloydWarshall() {
            free(distance[0]);
            free(distance);
            hipFree(deviceInput);
        }

        void SetDist(size_t i, size_t j, T c) { distance[i][j] = c; }
        T GetDist(size_t i, size_t j) { return distance[i][j]; }
        void Calc() {
            for (size_t k = 0; k < n; k++) {
                for (size_t i = 0; i < n; i++) {
                    for (size_t j = 0; j < n; j++) {
                        if (distance[i][k] + distance[k][j] < distance[i][j]) {
                            distance[i][j] = distance[i][k] + distance[k][j];
                        }
                    }
                }
            }
        }
        void DeviceCalc() {
            hipMemcpy(deviceInput, (void *) distance[0], n * n * sizeof(T), hipMemcpyDefault);
            for (size_t r = 1; r < n; r <<= 1) {
                Calc_<<<dim3((n + BS - 1) / BS, (n + BS - 1) / BS), dim3(BS, BS)>>>(n, deviceInput);
            }
            hipMemcpy((void *) distance[0], deviceInput, n * n * sizeof(T), hipMemcpyDefault);
            for (size_t i = 0; i < n; i++) { distance[i] = distance[0] + i * n; }
        }
    };

#undef BS
#undef inf
}; // namespace apsp
