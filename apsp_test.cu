#include "apsp.cu"
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>
using namespace std;

#define n (1024)
int d[n][n], res1[n][n], res2[n][n];

int main() {
    std::random_device rd;
    std::mt19937 gen(rd());

    apsp::FloydWarshall wf(n);

    std::chrono::high_resolution_clock::time_point t1, t2;
    std::chrono::duration<double> time_span;

    std::fstream in("in.txt", std::ios::out);
    std::fstream out1("out1.txt", std::ios::out);
    std::fstream out2("out2.txt", std::ios::out);

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) {
            d[i][j] = i == j ? 0 : gen() % 1000 + 1;
            in << d[i][j] << " ";
        }
        in << std::endl;
    }

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) { wf.SetDist(i, j, d[i][j]); }
    }

    hipDeviceSynchronize();
    t1 = std::chrono::high_resolution_clock::now();
    wf.DeviceCalc();
    hipDeviceSynchronize();
    t2 = std::chrono::high_resolution_clock::now();
    time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);
    std::cout << "GPU: Time: " << time_span.count() << "s" << std::endl;

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) {
            res1[i][j] = wf.GetDist(i, j);
            out1 << res1[i][j] << " ";
        }
        out1 << std::endl;
    }

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) { wf.SetDist(i, j, d[i][j]); }
    }

    t1 = std::chrono::high_resolution_clock::now();
    wf.Calc();
    t2 = std::chrono::high_resolution_clock::now();
    time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);
    std::cout << "CPU: Time: " << time_span.count() << "s" << std::endl;

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) {
            res2[i][j] = wf.GetDist(i, j);
            out2 << res2[i][j] << " ";
        }
        out2 << std::endl;
    }

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) {
            if (res1[i][j] != res2[i][j]) {
                std::cout << "Error: res[" << i << "][" << j << "]: " << res1[i][j] << " / "
                          << res2[i][j] << std::endl;
                return 1;
            }
        }
    }
    std::cout << "OK" << std::endl;
}
